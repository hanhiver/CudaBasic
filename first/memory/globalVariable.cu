#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "../common.h"

__device__ float devData;

__global__ void checkGlobalVariable()
{
    // display the original value.
    printf("Device: the value is %f\n", devData);

    // alter the value.
    devData *= 2.0f;
}

int main(void)
{
    // initialize the global variable. 
    float value = 3.14f;

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));
    printf("Host: copied %f to the global variable.\n", value);

    // invoke the kernel. 
    checkGlobalVariable <<<2, 2>>>();

    // copy the global variable back to the host. 
    CHECK(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
    printf("Host: the value changed by the kernel to %f\n", value);

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}


